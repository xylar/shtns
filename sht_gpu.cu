#include "hip/hip_runtime.h"

/* TODO
 * 1) try to store data for complex2real fft, and perform fft on host (less data to transfer)
 * 2) use static polar optimization (from constant memory ?)
 * 3) use a for loop in m-direction to re-use threads at larger m's.
 * 4) use double2 (double4 ?) vector types to avoid the use of shared memory to write mangled data.
 * 5) multi-stream / multi-gpu computation ?
 * 6) allow several variants, which may change occupancy for large sizes.
 */
 
// NOTE variables gridDim.x, blockIdx.x, blockDim.x, threadIdx.x, and warpSize are defined in device functions

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "sht_private.h"

#include <hipfft/hipfft.h>

// 256 for scalar SH_to_spat seems best on kepler.
#define THREADS_PER_BLOCK 256
// number of latitudes per thread:
#define NWAY 1
// number of l blocks to pre-compute
#define LSPAN 16
// the warp size is always 32 on cuda devices (up to Pascal at least)
#define WARPSZE 32
// maximum streams:
#define MAX_STRM 2

// adjustment for cuda
#undef SHT_L_RESCALE_FLY
#undef SHT_ACCURACY
#define SHT_L_RESCALE_FLY 1800
#define SHT_ACCURACY 1.0e-40

hipStream_t strm[MAX_STRM];

extern "C"
void* shtns_malloc(size_t size) {
    void* ptr = NULL;
    hipHostMalloc(&ptr, size);		// allocate pinned memory (for faster transfers !)
    return ptr;
}

extern "C"
void shtns_free(void* p) {
    hipHostFree(p);
}

/// On KEPLER, This kernel is fastest with THREADS_PER_BLOCK=256 and NWAY=1
template<int S> __global__ void
leg_m0(const double *al, const double *ct, const double *ql, double *q, const int llim, const int nlat_2)
{
    // im = 0
    const int it = blockDim.x * blockIdx.x + threadIdx.x;
    const int j = threadIdx.x;

    __shared__ double ak[THREADS_PER_BLOCK];
    __shared__ double qk[THREADS_PER_BLOCK/2];
    ak[j] = al[j];
    if ((j <= llim)&&(j<THREADS_PER_BLOCK/2)) qk[j] = ql[2*j];
    __syncthreads();

    int l = 0;
    int k = 0;	int kq = 0;
    double cost[NWAY];
    double y0[NWAY];    double y1[NWAY];
    double re[NWAY];    double ro[NWAY];

    for (int i=0; i<NWAY; i++) {
	cost[i] = (it+i<nlat_2) ? ct[it+i] : 0.0;
	y0[i] = ak[0];
	if (S==1) y0[i] *= rsqrt(1.0 - cost[i]*cost[i]);	// for vectors, divide by sin(theta)
    }
    for (int i=0; i<NWAY; i++) {
	re[i] = y0[i] * qk[0];
	y1[i] = y0[i] * ak[1] * cost[i];
    }
    for (int i=0; i<NWAY; i++) {
	ro[i] = y1[i] * qk[1];
    }
    al+=2;    l+=2;	k+=2;	kq+=2;
    while(l<llim) {
	if (k+6 >= THREADS_PER_BLOCK) {
	    __syncthreads();
	    ak[j] = al[j];
	    if ((j <= llim)&&(j<THREADS_PER_BLOCK/2)) qk[j] = ql[2*(l+j)];
	    k=0;	kq=0;
    	    __syncthreads();
	}
	for (int i=0; i<NWAY; i++)	y0[i]  = ak[k+1]*cost[i]*y1[i] + ak[k]*y0[i];
	for (int i=0; i<NWAY; i++)	re[i] += y0[i] * qk[kq];
	for (int i=0; i<NWAY; i++)	y1[i]  = ak[k+3]*cost[i]*y0[i] + ak[k+2]*y1[i];
	for (int i=0; i<NWAY; i++)	ro[i] += y1[i] * qk[kq+1];
	al+=4;	l+=2;	k+=4;	kq+=2;
    }
    if (l==llim) {
	for (int i=0; i<NWAY; i++)	y0[i]  = ak[k+1]*cost[i]*y1[i] + ak[k]*y0[i];
	for (int i=0; i<NWAY; i++)	re[i] += y0[i] * qk[kq];
    }

    for (int i=0; i<NWAY; i++) {
	if (it+i < nlat_2) {
	    q[it+i] = re[i]+ro[i];
	    q[nlat_2*2-1-(it+i)] = re[i]-ro[i];
	}
    }
/*
    if (it < nlat_2) {
        int l = 0;
	double cost = ct[it];
        double y0 = al[0];
        double re = y0 * ql[0];
        double y1 = y0 * al[1] * cost;
        double ro = y1 * ql[1];
        al+=2;    l+=2;
        while(l<llim) {
            y0  = al[1]*(cost*y1) + al[0]*y0;
            re += y0 * ql[l];
            y1  = al[3]*(cost*y0) + al[2]*y1;
            ro += y1 * ql[l+1];
            al+=4;	l+=2;
        }
        if (l==llim) {
            y0  = al[1]*cost*y1 + al[0]*y0;
            re += y0 * ql[l];
        }

        q[it] = re+ro;
        q[nlat_2*2-1-it] = re-ro;
    }
    */
}

__inline__ __device__
void warp_reduce_add_4(double& re, double& ro, double& ie, double& io) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
    re += __shfl_down(re, offset);
    ro += __shfl_down(ro, offset);
    ie += __shfl_down(ie, offset);
    io += __shfl_down(io, offset);
  }
}

__inline__ __device__
void warp_reduce_add_2(double& ev, double& od) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
    ev += __shfl_down(ev, offset);
    od += __shfl_down(od, offset);
  }
}

__inline__ __device__
void warp_reduce_add(double& ev) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
    ev += __shfl_down(ev, offset);
  }
}

#if (__CUDACC_VER_MAJOR__ < 8) || ( defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600 )
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
	old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif


/// THREADS_PER_BLOCK/LSPAN must be a power of 2 and <= WARPSZE
/// LSPAN must be a multiple of 2.
template<int S> __global__ void
ileg_m0(const double *al, const double *ct, const double *q, double *ql, const int llim, const int nlat_2)
{
    // im = 0
    const int it = (blockDim.x * blockIdx.x + threadIdx.x)*NWAY;
    const int j = threadIdx.x;

    __shared__ double ak[2*LSPAN+2];	// cache
    __shared__ double yl[LSPAN][THREADS_PER_BLOCK+1];	// padding to avoid bank conflicts
    __shared__ double reo[2][THREADS_PER_BLOCK+1];	// padding to avoid bank conflicts
    double cost, y0, y1;

    y0 = (it < nlat_2) ? q[it] : 0.0;		// north
    y1 = (it < nlat_2) ? q[nlat_2*2-1 - it] : 0.0;	// south
    reo[0][j] = y0+y1;				// even
    reo[1][j] = y0-y1;		// odd

    if (j < 2*LSPAN+2) ak[j] = al[j];
    #if THREADS_PER_BLOCK > WARPSZE
    __syncthreads();
    #endif

    int l = 0;
    if (it < nlat_2) {
	y0 = ct[it + nlat_2];		// weights are stored just after ct.
	cost = ct[it];
    } else {
	y0 = 0.0;	cost = 0.0;
    }
    if (S==1) y0 *= rsqrt(1.0 - cost*cost);	// for vectors, divide by sin(theta)
    y0 *= ak[0];
    y1 = y0 * ak[1] * cost;
    yl[0][j] = y0;
    yl[1][j] = y1;
    al+=2;
    while (l < llim) {
	for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
	    yl[k][j]     = y0;
	    y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
	    yl[k+1][j] = y1;
	    y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
	    al += 4;
	}

	#if THREADS_PER_BLOCK > WARPSZE
	__syncthreads();
	#endif
	double qll = 0.0;	// accumulator
	// now re-assign each thread an l (transpose)
	const int ll = j / (THREADS_PER_BLOCK/LSPAN);
	for (int i=0; i<THREADS_PER_BLOCK; i+= THREADS_PER_BLOCK/LSPAN) {
	    int it = j % (THREADS_PER_BLOCK/LSPAN) + i;
	    qll += reo[ll&1][it] * yl[ll][it];
	}
	// reduce_add within same l must be in same warp too:
	#if THREADS_PER_BLOCK/LSPAN > WARPSZE
	    #error "THREADS_PER_BLOCK/LSPAN > WARPSZE"
	#endif
	for (int ofs = THREADS_PER_BLOCK/(LSPAN*2); ofs > 0; ofs>>=1) {
	    qll += __shfl_down(qll, ofs, THREADS_PER_BLOCK/LSPAN);
	}
	if ( ((j % (THREADS_PER_BLOCK/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
	    if (nlat_2 <= THREADS_PER_BLOCK) {		// do we need atomic add or not ?
		ql[2*(l+ll)] = qll;
	    } else {
		atomicAdd(ql+2*(l+ll), qll);		// VERY slow atomic add on Kepler.
	    }
	}
	if (j<2*LSPAN) ak[j+2] = al[j];
	#if THREADS_PER_BLOCK > WARPSZE
	__syncthreads();
	#endif
	l+=LSPAN;
    }
}


/** \internal convert from vector SH to scalar SH
    Vlm =  st*d(Slm)/dtheta + I*m*Tlm
    Wlm = -st*d(Tlm)/dtheta + I*m*Slm
*/
__global__ void
sphtor2scal_gpu(const double *mx, const double *slm, const double *tlm, double *vlm, double *wlm, const int llim, const int lmax, const int mres)
{
    const int ll = (blockDim.x-4) * blockIdx.x + threadIdx.x - 2;		// = 2*l + ((imag) ? 1 : 0)
    const int j = threadIdx.x;
    const int im = blockIdx.y;

    __shared__ double sl[THREADS_PER_BLOCK];
    __shared__ double tl[THREADS_PER_BLOCK];
    __shared__ double M[THREADS_PER_BLOCK];

    const int m = __mul24(im, mres);
    //int ofs = im*(2*(lmax+1) -m + mres);
//    const int xchg = 1 - 2*(ll&1);	// +1 for real and -1 for imag
    const int xchg = ll - (ll^1);	// -1 for real and +1 for imag
    const int ofs = __mul24( im, (((lmax+1)<<1) -m + mres) ) + ll;

    if ( (ll >= 0) && (ll < 2*(llim+1-m)) ) {
	M[j] = mx[ofs];
	sl[j] = slm[ofs];
	tl[j] = tlm[ofs];
    } else {
	M[j] = 0.0;
	sl[j] = 0.0;
	tl[j] = 0.0;
    }
    const double mimag = im * mres * (ll - (ll^1));

    __syncthreads();

//    if ((j>=2) && (j<THREADS_PER_BLOCK-2) && (ll < 2*(llim+2-m))) {
    if ((j<THREADS_PER_BLOCK-4) && (ll < 2*(llim+1-m))) {
	double ml = M[2*(j>>1)+1];
	double mu = M[2*(j>>1)+2];
	double v = mimag*tl[(j+2)^1]  +  (ml*sl[j] + mu*sl[j+4]);
	double w = mimag*sl[(j+2)^1]  -  (ml*tl[j] + mu*tl[j+4]);
	vlm[ofs+2*im+2] = v;
	wlm[ofs+2*im+2] = w;
    }
}

/** \internal convert from 2 scalar SH to vector SH
    Slm = - (I*m*Wlm + MX*Vlm) / (l*(l+1))
    Tlm = - (I*m*Vlm - MX*Wlm) / (l*(l+1))
**/
__global__ void
scal2sphtor_gpu(const double *mx, const double *vlm, const double *wlm, double *slm, double *tlm, const int llim, const int lmax, const int mres)
{
    const int ll = (blockDim.x-4) * blockIdx.x + threadIdx.x - 2;		// = 2*l + ((imag) ? 1 : 0)
    const int j = threadIdx.x;
    const int im = blockIdx.y;

    __shared__ double vl[THREADS_PER_BLOCK];
    __shared__ double wl[THREADS_PER_BLOCK];
    __shared__ double M[THREADS_PER_BLOCK];

    const int m = __mul24(im, mres);
    //const int xchg = 1 - 2*(j&1);	// +1 for real and -1 for imag
    const int xchg = (j^1) - j;		// +1 for real and -1 for imag
    int ofs = im*(2*(lmax+1) -m + mres)  + ll;
    //int ofs = __mul24( im, (((lmax+1)<<1) -m + mres) )  + ll;

    if ( (ll >= 0) && (ll < 2*(llim+1-m)) ) {
	M[j] = mx[ofs];
    } else M[j] = 0.0;

    if ( (ll >= 0) && (ll < 2*(llim+2-m)) ) {
	vl[j] = vlm[ofs+2*im];
	wl[j] = wlm[ofs+2*im];
    } else {
	vl[j] = 0.0;
	wl[j] = 0.0;
    }

    int ell = (ll>>1) + m + 1;		// +1 because we shift below

    __syncthreads();

//    if ((j>=2) && (j<THREADS_PER_BLOCK-2) && (ll < 2*(llim+1-m))) {
    if (j<THREADS_PER_BLOCK-4) {
	if ((ell <= llim) && (ell>0)) {
	    const double mimag = im * xchg*mres;
	    double ll_1 = 1.0 / __mul24(ell,ell+1);
	    double ml = M[2*(j>>1)+1];
	    double mu = M[2*(j>>1)+2];
	    double s = mimag*wl[(j+2)^1]  -  (ml*vl[j] + mu*vl[j+4]);
	    double t = mimag*vl[(j+2)^1]  +  (ml*wl[j] + mu*wl[j+4]);
	    slm[ofs+2] = s * ll_1;
	    tlm[ofs+2] = t * ll_1;
	} else if (ell <= lmax) {	// fill with zeros up to lmax (and l=0 too).
	    slm[ofs+2] = 0.0;
	    tlm[ofs+2] = 0.0;
	}
    }
}


/// requirements : blockSize must be 1 in the y-direction and THREADS_PER_BLOCK in the x-direction.
/// llim MUST BE <= 1800
/// S can only be 0 (for scalar) or 1 (for spin 1 / vector)
template<int S> __global__ void
leg_m_lowllim(const double *al, const double *ct, const double *ql, double *q, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi)
{
    const int it = blockDim.x * blockIdx.x + threadIdx.x;
    const int im = blockIdx.y;
    const int j = threadIdx.x;
    const int m_inc = 2*nlat_2;
    const int k_inc = 1;

    // two arrays in shared memory of size blockDim.x :
    extern __shared__ double ak[];
    double* const qk = ak + blockDim.x;

    const double cost = (it < nlat_2) ? ct[it] : 0.0;

    if (im==0) {
	ak[j] = al[j];
	if (j<2*(llim+1)) qk[j] = ql[j];
	__syncthreads();
	int l = 0;
	int ka = 0;	int kq = 0;
	double y0 = ak[0];
	if (S==1) y0 *= rsqrt(1.0 - cost*cost);	// for vectors, divide by sin(theta)
	double re = y0 * qk[0];
	double y1 = y0 * ak[1] * cost;
	double ro = y1 * qk[2];
	al+=2;    l+=2;		ka+=2;	kq+=2;
	while(l<llim) {
	    if (ka+6 >= blockDim.x) {
		__syncthreads();  
		ak[j] = al[j];
		qk[j] = ql[2*l+j];
		ka=0;	kq=0;
		__syncthreads();
	    }
	    y0  = ak[ka+1]*cost*y1 + ak[ka]*y0;
	    re += y0 * qk[2*kq];
	    y1  = ak[ka+3]*cost*y0 + ak[ka+2]*y1;
	    ro += y1 * qk[2*kq+2];
	    al+=4;	l+=2;	  ka+=4;    kq+=2;
	}
	if (l==llim) {
	    y0  = ak[ka+1]*cost*y1 + ak[ka]*y0;
	    re += y0 * qk[2*kq];
	}
	if (it<nlat_2) {
	    // store mangled for complex fft
	    q[it*k_inc] = re+ro;
	    q[(nlat_2*2-1-it)*k_inc] = re-ro;
	}
    } else { 	// m>0
	double rer,ror, rei, roi, y0, y1;
	int m = im*mres;
	int l = (im*(2*(lmax+1)-(m+mres)))>>1;
	al += 2*(l+m);
	ql += 2*(l + S*im);	// allow vector transforms where llim = lmax+1

	y1 = sqrt(1.0 - cost*cost);		// y1 = sin(theta)
	ak[j] = al[j];
	qk[j] = ql[2*m+j];

	ror = 0.0;		roi = 0.0;
	rer = 0.0;		rei = 0.0;
	y0 = 1.0;
	l = m - S;
	do {		// sin(theta)^(m-S)
	    if (l&1) y0 *= y1;
	    y1 *= y1;
	} while(l >>= 1);
	
	__syncthreads();
	y0 *= ak[0];
	y1 = ak[1]*y0*cost;

	int ka = 2;
	l=m;		al+=2;
	int kq = 0;

	while (l<llim) {	// compute even and odd parts
	    if (2*kq+6 > blockDim.x) {
		__syncthreads();
		ak[j] = al[j];
		qk[j] = ql[2*l+j];
		ka=0;	kq=0;
		__syncthreads();
	    }
	    rer += y0 * qk[2*kq];	// real
	    rei += y0 * qk[2*kq+1];	// imag
	    y0 = ak[ka+1]*(cost*y1) + ak[ka]*y0;
	    ror += y1 * qk[2*kq+2];	// real
	    roi += y1 * qk[2*kq+3];	// imag
	    y1 = ak[ka+3]*(cost*y0) + ak[ka+2]*y1;
	    l+=2;	al+=4;	 ka+=4;	  kq+=2;
	}
	if (l==llim) {
	    rer += y0 * qk[2*kq];
	    rei += y0 * qk[2*kq+1];
	}

	/// store mangled for complex fft
	double nr = rer+ror;
	double sr = rer-ror;
	const double sgn = 1 - 2*(j&1);
	rei = __shfl_xor(rei, 1);
	roi = __shfl_xor(roi, 1);
	double nix = sgn*(rei+roi);
	double six = sgn*(rei-roi);
	if (it < nlat_2) {
	    q[im*m_inc + it*k_inc]                     = nr - nix;
	    q[(nphi-im)*m_inc + it*k_inc]              = nr + nix;
	    q[im*m_inc + (nlat_2*2-1-it)*k_inc]        = sr + six;
	    q[(nphi-im)*m_inc + (nlat_2*2-1-it)*k_inc] = sr - six;
	}
    }
}

/// requirements : blockSize must be 1 in the y-direction and THREADS_PER_BLOCK in the x-direction.
/// llim can be arbitrarily large (> 1800)
template<int S> __global__ void
leg_m_highllim(const double *al, const double *ct, const double *ql, double *q, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi)
{
    const int it = blockDim.x * blockIdx.x + threadIdx.x;
    const int im = blockIdx.y;
    const int j = threadIdx.x;
    const int m_inc = 2*nlat_2;
    const int k_inc = 1;

    __shared__ double ak[THREADS_PER_BLOCK];	// cache
    __shared__ double qk[THREADS_PER_BLOCK];

    const double cost = (it < nlat_2) ? ct[it] : 0.0;

    if (im==0) {
	int l = 0;
	double y0 = al[0];
	if (S==1) y0 *= rsqrt(1.0 - cost*cost);
	double re = y0 * ql[0];
	double y1 = y0 * al[1] * cost;
	double ro = y1 * ql[2];
	al+=2;    l+=2;
	while(l<llim) {
	    y0  = al[1]*(cost*y1) + al[0]*y0;
	    re += y0 * ql[2*l];
	    y1  = al[3]*(cost*y0) + al[2]*y1;
	    ro += y1 * ql[2*l+2];
	    al+=4;	l+=2;
	}
	if (l==llim) {
	    y0  = al[1]*cost*y1 + al[0]*y0;
	    re += y0 * ql[2*l];
	}
	if (it < nlat_2) {
	    // store mangled for complex fft
	    q[it*k_inc] = re+ro;
	    q[(nlat_2*2-1-it)*k_inc] = re-ro;
	}
    } else { 	// m>0
	int m = im*mres;
	int l = (im*(2*(lmax+1)-(m+mres)))>>1;
	al += 2*(l+m);
	ql += 2*(l + S*im);
	double rer,ror, rei,roi, y0, y1;
	ror = 0.0;	roi = 0.0;
	rer = 0.0;	rei = 0.0;
	y1 = sqrt(1.0 - cost*cost);	// sin(theta)
	if (__any(m - llim*y1 <= max(50,llim/200))) {		// polar optimization (see Reinecke 2013), avoiding warp divergence
	    y0 = 1.0;	// y0
	    l = m - S;
	    int ny = 0;
	    int nsint = 0;
	    do {		// sin(theta)^(m-S)		(use rescaling to avoid underflow)
		if (l&1) {
		    y0 *= y1;
		    ny += nsint;
		    if (__any(y0 < (SHT_ACCURACY+1.0/SHT_SCALE_FACTOR))) {		// avoid warp divergence
			ny--;
			y0 *= SHT_SCALE_FACTOR;
		    }
		}
		y1 *= y1;
		nsint += nsint;
		if (__any(y1 < 1.0/SHT_SCALE_FACTOR)) {		// avoid warp divergence
		    nsint--;
		    y1 *= SHT_SCALE_FACTOR;
		}
	    } while(l >>= 1);
	    y0 *= al[0];
	    y1 = 0.0;
//	    y1 = al[1]*y0*cost;

	    l=m;	int ka = WARPSZE;
	    const int ofs = j & 0xFFE0;

	    while ( __all(ny<0) && (l<llim) ) {
		if (ka+4 >= WARPSZE) {
		    ak[j] = al[(j&31)];
		    ka=0;
		}
		y1 = ak[ka+1+ofs]*cost*y0 + ak[ka+ofs]*y1;
		y0 = ak[ka+3+ofs]*cost*y1 + ak[ka+2+ofs]*y0;
		l+=2;	al+=4;	ka+=4;
		if (fabs(y1) > SHT_ACCURACY*SHT_SCALE_FACTOR + 1.0)
		{	// rescale when value is significant
		    ++ny;
		    y0 *= 1.0/SHT_SCALE_FACTOR;
		    y1 *= 1.0/SHT_SCALE_FACTOR;
		}
	    }
	    
	    ka = WARPSZE;
	    while (l<llim) {
		if (ka+4 >= WARPSZE) {		// cache coefficients
		    ak[j] = al[(j&31)];
		    qk[j] = ql[2*l+(j&31)];
		    ka = 0;
		}
		y1 = ak[ka+1+ofs]*cost*y0 + ak[ka+ofs]*y1;
		if (ny==0) {
		    rer += y0 * qk[ka+ofs];	// real
		    rei += y0 * qk[ka+1+ofs];	// imag
		    ror += y1 * qk[ka+2+ofs];	// real
		    roi += y1 * qk[ka+3+ofs];	// imag
		}
		else if (fabs(y0) > SHT_ACCURACY*SHT_SCALE_FACTOR + 1.0)
		{	// rescale when value is significant
		    ++ny;
		    y0 *= 1.0/SHT_SCALE_FACTOR;
		    y1 *= 1.0/SHT_SCALE_FACTOR;
		}
		l+=2;	al+=4;
		y0 = ak[ka+3+ofs]*cost*y1 + ak[ka+2+ofs]*y0;
		ka+=4;
	    }
	    if ((l==llim) && (ny==0)) {
		rer += y0 * ql[2*l];
		rei += y0 * ql[2*l+1];
	    }
	}

	/// store mangled for complex fft
	double nr = rer+ror;
	double sr = rer-ror;
	const double sgn = 1 - 2*(j&1);
	rei = __shfl_xor(rei, 1);
	roi = __shfl_xor(roi, 1);
	double nix = sgn*(rei+roi);
	double six = sgn*(rei-roi);
	if (it < nlat_2) {
	    q[im*m_inc + it*k_inc]                     = nr - nix;
	    q[(nphi-im)*m_inc + it*k_inc]              = nr + nix;
	    q[im*m_inc + (nlat_2*2-1-it)*k_inc]        = sr + six;
	    q[(nphi-im)*m_inc + (nlat_2*2-1-it)*k_inc] = sr - six;
	}
    }
}


template<int S> __global__ void
ileg_m_lowllim(const double *al, const double *ct, const double *q, double *ql, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi)
{
    const int it = (blockDim.x * blockIdx.x + threadIdx.x)*NWAY;
    const int j = threadIdx.x;
    const int im = blockIdx.y;
    const int m_inc = 2*nlat_2;
//    const int k_inc = 1;

    __shared__ double ak[2*LSPAN+2];	// cache
    __shared__ double yl[LSPAN*THREADS_PER_BLOCK];
    __shared__ double reo[4*THREADS_PER_BLOCK];
    const int l_inc = THREADS_PER_BLOCK;
    const double cost = (it < nlat_2) ? ct[it] : 0.0;
    double y0, y1;


    if (im == 0) {
	if (j < 2*LSPAN+2) ak[j] = al[j];
	#if THREADS_PER_BLOCK > WARPSZE
	__syncthreads();
	#endif
	y0 = (it < nlat_2) ? q[it] : 0.0;		// north
	y1 = (it < nlat_2) ? q[nlat_2*2-1 - it] : 0.0;	// south
	reo[j] = y0+y1;				// even
	reo[THREADS_PER_BLOCK +j] = y0-y1;		// odd

	int l = 0;
	y0 = (it < nlat_2) ? ct[it + nlat_2] : 0.0;		// weights are stored just after ct.
	if (S==1) y0 *= rsqrt(1.0 - cost*cost);
	y0 *= ak[0];
	y1 = y0 * ak[1] * cost;
	yl[j] = y0;
	yl[l_inc +j] = y1;
	al+=2;
	while (l <= llim) {
	    for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
		yl[k*l_inc +j]     = y0;
		y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
		yl[(k+1)*l_inc +j] = y1;
		y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
		al += 4;
	    }
	    #if THREADS_PER_BLOCK > WARPSZE
	    __syncthreads();
	    #endif
	    double qll = 0.0;	// accumulator
	    // now re-assign each thread an l (transpose)
	    const int ll = j / (THREADS_PER_BLOCK/LSPAN);
	    for (int i=0; i<THREADS_PER_BLOCK; i+= THREADS_PER_BLOCK/LSPAN) {
		int it = j % (THREADS_PER_BLOCK/LSPAN) + i;
		qll += reo[(ll&1)*THREADS_PER_BLOCK +it] * yl[ll*l_inc +it];
	    }
	    // reduce_add within same l must be in same warp too:
	    #if THREADS_PER_BLOCK/LSPAN > WARPSZE
		#error "THREADS_PER_BLOCK/LSPAN > WARPSZE"
	    #endif
	    for (int ofs = THREADS_PER_BLOCK/(LSPAN*2); ofs > 0; ofs>>=1) {
		qll += __shfl_down(qll, ofs, THREADS_PER_BLOCK/LSPAN);
	    }
	    if ( ((j % (THREADS_PER_BLOCK/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
		if (nlat_2 <= THREADS_PER_BLOCK) {		// do we need atomic add or not ?
		    ql[2*(l+ll)] = qll;
		} else {
		    atomicAdd(ql+2*(l+ll), qll);		// VERY slow atomic add on Kepler.
		}
	    }
	    if (j<2*LSPAN) ak[j+2] = al[j];
	    #if THREADS_PER_BLOCK > WARPSZE
	    __syncthreads();
	    #endif
	    l+=LSPAN;
	}
    } else {	// im > 0
	int m = im*mres;
	int l = (im*(2*(lmax+1)-(m+mres)))>>1;
	al += 2*(l+m);
	ql += 2*(l + S*im);	// allow vector transforms where llim = lmax+1

	if (j < 2*LSPAN+2) ak[j] = al[j];
	#if THREADS_PER_BLOCK > WARPSZE
	__syncthreads();
	#endif
	const double sgn = 2*(j&1) - 1;	// -/+
	y0    = (it < nlat_2) ? q[im*m_inc + it] : 0.0;		// north imag (ani)
	double qer    = (it < nlat_2) ? q[(nphi-im)*m_inc + it] : 0.0;	// north real (an)
	y1    = (it < nlat_2) ? q[im*m_inc + nlat_2*2-1-it] : 0.0;	// south imag (asi)
	double qor    = (it < nlat_2) ? q[(nphi-im)*m_inc + nlat_2*2-1-it] : 0.0;	// south real (as)
	double qei = y0-qer;		qer += y0;		// ani = -qei[lane+1],   bni = qei[lane-1]
	double qoi = y1-qor;		qor += y1;		// bsi = -qoi[lane-1],   asi = qoi[lane+1];
	y0 = __shfl_xor(qei, 1);	// exchange between adjacent lanes.
	y1 = __shfl_xor(qoi, 1);
	reo[j] 			    = qer + qor;	// rer
	reo[THREADS_PER_BLOCK +j]   = qer - qor;	// ror
	reo[2*THREADS_PER_BLOCK +j] = sgn*(y0 - y1);	// rei
	reo[3*THREADS_PER_BLOCK +j] = sgn*(y0 + y1);	// roi
    	
	y1 = sqrt(1.0 - cost*cost);	// sin(theta)

	    y0 = 0.5 * ak[0];	// y0
	    l = m - S;
	    do {		// sin(theta)^(m-S)
		if (l&1) y0 *= y1;
		y1 *= y1;
	    } while(l >>= 1);
	    if (it < nlat_2)     y0 *= ct[it + nlat_2];		// include quadrature weights.
	    y1 = ak[1]*y0*cost;

	    l=m;		al+=2;
	    while (l <= llim) {
		for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
		    yl[k*l_inc +j]     = y0;
		    y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
		    yl[(k+1)*l_inc +j] = y1;
		    y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
		    al += 4;
		}

		#if THREADS_PER_BLOCK > WARPSZE
		__syncthreads();
		#endif
		double qlri = 0.0;	// accumulator
		// now re-assign each thread an l (transpose)
		const int ll = j / (THREADS_PER_BLOCK/LSPAN);
		const int ri = j / (THREADS_PER_BLOCK/(2*LSPAN)) % 2;	// real (0) or imag (1)
		for (int i=0; i<THREADS_PER_BLOCK; i+= THREADS_PER_BLOCK/(2*LSPAN)) {
		    int it = j % (THREADS_PER_BLOCK/(2*LSPAN)) + i;
		    qlri += reo[((ll&1)+2*ri)*THREADS_PER_BLOCK +it]   * yl[ll*l_inc +it];
		}
		// reduce_add within same l must be in same warp too:
		#if THREADS_PER_BLOCK/(2*LSPAN) > WARPSZE
		    #error "THREADS_PER_BLOCK/(2*LSPAN) > WARPSZE"
		#endif
		for (int ofs = THREADS_PER_BLOCK/(LSPAN*4); ofs > 0; ofs>>=1) {
		    qlri += __shfl_down(qlri, ofs, THREADS_PER_BLOCK/(LSPAN*2));
		}
		if ( ((j % (THREADS_PER_BLOCK/(2*LSPAN))) == 0) && ((l+ll)<=llim) ) {	// write result
		    if (nlat_2 <= THREADS_PER_BLOCK) {		// do we need atomic add or not ?
			ql[2*(l+ll)+ri]   = qlri;
		    } else {
			atomicAdd(ql+2*(l+ll)+ri, qlri);		// VERY slow atomic add on Kepler.
		    }
		}
		if (j<2*LSPAN) ak[j+2] = al[j];
		#if THREADS_PER_BLOCK > WARPSZE
		__syncthreads();
		#endif
		l+=LSPAN;
	    }
    }
}


template<int S> __global__ void
ileg_m_highllim(const double *al, const double *ct, const double *q, double *ql, const int llim, const int nlat_2, const int lmax, const int mres, const int nphi)
{
    const int it = (blockDim.x * blockIdx.x + threadIdx.x)*NWAY;
    const int j = threadIdx.x;
    const int im = blockIdx.y;
    const int m_inc = 2*nlat_2;
//    const int k_inc = 1;

    __shared__ double ak[2*LSPAN+2];	// cache
    __shared__ double yl[LSPAN*THREADS_PER_BLOCK];
    __shared__ double reo[4*THREADS_PER_BLOCK];
    const int l_inc = THREADS_PER_BLOCK;
    const double cost = (it < nlat_2) ? ct[it] : 0.0;
    double y0, y1;


    if (im == 0) {
	if (j < 2*LSPAN+2) ak[j] = al[j];
	#if THREADS_PER_BLOCK > WARPSZE
	__syncthreads();
	#endif
	y0 = (it < nlat_2) ? q[it] : 0.0;		// north
	y1 = (it < nlat_2) ? q[nlat_2*2-1 - it] : 0.0;	// south
	reo[j] = y0+y1;				// even
	reo[THREADS_PER_BLOCK +j] = y0-y1;		// odd

	int l = 0;
	y0 = (it < nlat_2) ? ct[it + nlat_2] : 0.0;		// weights are stored just after ct.
	if (S==1) y0 *= rsqrt(1.0 - cost*cost);
	y0 *= ak[0];
	y1 = y0 * ak[1] * cost;
	yl[j] = y0;
	yl[l_inc +j] = y1;
	al+=2;
	while (l <= llim) {
	    for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
		yl[k*l_inc +j]     = y0;
		y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
		yl[(k+1)*l_inc +j] = y1;
		y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
		al += 4;
	    }
	    #if THREADS_PER_BLOCK > WARPSZE
	    __syncthreads();
	    #endif
	    double qll = 0.0;	// accumulator
	    // now re-assign each thread an l (transpose)
	    const int ll = j / (THREADS_PER_BLOCK/LSPAN);
	    for (int i=0; i<THREADS_PER_BLOCK; i+= THREADS_PER_BLOCK/LSPAN) {
		int it = j % (THREADS_PER_BLOCK/LSPAN) + i;
		qll += reo[(ll&1)*THREADS_PER_BLOCK +it] * yl[ll*l_inc +it];
	    }
	    // reduce_add within same l must be in same warp too:
	    #if THREADS_PER_BLOCK/LSPAN > WARPSZE
		#error "THREADS_PER_BLOCK/LSPAN > WARPSZE"
	    #endif
	    for (int ofs = THREADS_PER_BLOCK/(LSPAN*2); ofs > 0; ofs>>=1) {
		qll += __shfl_down(qll, ofs, THREADS_PER_BLOCK/LSPAN);
	    }
	    if ( ((j % (THREADS_PER_BLOCK/LSPAN)) == 0) && ((l+ll)<=llim) ) {	// write result
		if (nlat_2 <= THREADS_PER_BLOCK) {		// do we need atomic add or not ?
		    ql[2*(l+ll)] = qll;
		} else {
		    atomicAdd(ql+2*(l+ll), qll);		// VERY slow atomic add on Kepler.
		}
	    }
	    if (j<2*LSPAN) ak[j+2] = al[j];
	    #if THREADS_PER_BLOCK > WARPSZE
	    __syncthreads();
	    #endif
	    l+=LSPAN;
	}
    } else {	// im > 0
	int m = im*mres;
	int l = (im*(2*(lmax+1)-(m+mres)))>>1;
	al += 2*(l+m);
	ql += 2*(l + S*im);	// allow vector transforms where llim = lmax+1

	if (j < 2*LSPAN+2) ak[j] = al[j];
	#if THREADS_PER_BLOCK > WARPSZE
	__syncthreads();
	#endif
	const double sgn = 2*(j&1) - 1;	// -/+
	y0    = (it < nlat_2) ? q[im*m_inc + it] : 0.0;		// north imag (ani)
	double qer    = (it < nlat_2) ? q[(nphi-im)*m_inc + it] : 0.0;	// north real (an)
	y1    = (it < nlat_2) ? q[im*m_inc + nlat_2*2-1-it] : 0.0;	// south imag (asi)
	double qor    = (it < nlat_2) ? q[(nphi-im)*m_inc + nlat_2*2-1-it] : 0.0;	// south real (as)
	double qei = y0-qer;		qer += y0;		// ani = -qei[lane+1],   bni = qei[lane-1]
	double qoi = y1-qor;		qor += y1;		// bsi = -qoi[lane-1],   asi = qoi[lane+1];
	y0 = __shfl_xor(qei, 1);	// exchange between adjacent lanes.
	y1 = __shfl_xor(qoi, 1);
	reo[j] 			    = qer + qor;	// rer
	reo[THREADS_PER_BLOCK +j]   = qer - qor;	// ror
	reo[2*THREADS_PER_BLOCK +j] = sgn*(y0 - y1);	// rei
	reo[3*THREADS_PER_BLOCK +j] = sgn*(y0 + y1);	// roi
    	
	y1 = sqrt(1.0 - cost*cost);	// sin(theta)

	    y0 = 0.5;	// y0
	    l = m - S;
	    int ny = 0;
	    int nsint = 0;
	    do {		// sin(theta)^(m-S)		(use rescaling to avoid underflow)
		if (l&1) {
		    y0 *= y1;
		    ny += nsint;
		    // the use of __any leads to wrong results. On KEPLER it is also slower.
//		    if (__any(y0 < (SHT_ACCURACY+1.0/SHT_SCALE_FACTOR))) {		// avoid warp divergence
		    if (y0 < (SHT_ACCURACY+1.0/SHT_SCALE_FACTOR)) {
			ny--;
			y0 *= SHT_SCALE_FACTOR;
		    }
		}
		y1 *= y1;
		nsint += nsint;
//		if (__any(y1 < 1.0/SHT_SCALE_FACTOR)) {		// avoid warp divergence
		if (y1 < 1.0/SHT_SCALE_FACTOR) {
		    nsint--;
		    y1 *= SHT_SCALE_FACTOR;
		}
	    } while(l >>= 1);
	    y0 *= ak[0];
	    if (it < nlat_2)     y0 *= ct[it + nlat_2];		// include quadrature weights.
	    y1 = ak[1]*y0*cost;


	    l=m;		al+=2;
	    while (l <= llim) {
		for (int k=0; k<LSPAN; k+=2) {		// compute a block of the matrix, write it in shared mem.
		    yl[k*l_inc +j]     = (ny==0) ? y0 : 0.0;
		    y0 = ak[2*k+3]*cost*y1 + ak[2*k+2]*y0;
		    yl[(k+1)*l_inc +j] = (ny==0) ? y1 : 0.0;
		    y1 = ak[2*k+5]*cost*y0 + ak[2*k+4]*y1;
		    if (ny<0) {
//			if (__any(fabs(y0) > SHT_ACCURACY*SHT_SCALE_FACTOR + 1.0))
			if (fabs(y0) > SHT_ACCURACY*SHT_SCALE_FACTOR + 1.0)
			{	// rescale when value is significant
			    ++ny;
			    y0 *= 1.0/SHT_SCALE_FACTOR;
			    y1 *= 1.0/SHT_SCALE_FACTOR;
			}
		    }
		    al += 4;
		}

		#if THREADS_PER_BLOCK > WARPSZE
		__syncthreads();
		#endif
		double qlri = 0.0;	// accumulator
		// now re-assign each thread an l (transpose)
		const int ll = j / (THREADS_PER_BLOCK/LSPAN);
		const int ri = j / (THREADS_PER_BLOCK/(2*LSPAN)) % 2;	// real (0) or imag (1)
		if (ll+l <= llim) {
		    for (int i=0; i<THREADS_PER_BLOCK; i+= THREADS_PER_BLOCK/(2*LSPAN)) {
			int it = j % (THREADS_PER_BLOCK/(2*LSPAN)) + i;
			qlri += reo[((ll&1)+2*ri)*THREADS_PER_BLOCK +it]   * yl[ll*l_inc +it];
		    }
		}
		// reduce_add within same l must be in same warp too:
		#if THREADS_PER_BLOCK/(2*LSPAN) > WARPSZE
		    #error "THREADS_PER_BLOCK/(2*LSPAN) > WARPSZE"
		#endif
		for (int ofs = THREADS_PER_BLOCK/(LSPAN*4); ofs > 0; ofs>>=1) {
		    qlri += __shfl_down(qlri, ofs, THREADS_PER_BLOCK/(LSPAN*2));
		}
		if ( ((j % (THREADS_PER_BLOCK/(2*LSPAN))) == 0) && ((l+ll)<=llim) ) {	// write result
		    if (nlat_2 <= THREADS_PER_BLOCK) {		// do we need atomic add or not ?
			ql[2*(l+ll)+ri]   = qlri;
		    } else {
			atomicAdd(ql+2*(l+ll)+ri, qlri);		// VERY slow atomic add on Kepler.
		    }
		}
		if (j<2*LSPAN) ak[j+2] = al[j];
		#if THREADS_PER_BLOCK > WARPSZE
		__syncthreads();
		#endif
		l+=LSPAN;
	    }
    }
}





extern "C"
int cushtns_init_gpu(shtns_cfg shtns)
{
    hipError_t err = hipSuccess;
    const long nlm = shtns->nlm;
    const long nlat_2 = shtns->nlat_2;

    double *d_alm = NULL;
    double *d_ct  = NULL;
    double *d_qlm = NULL;
    double *d_q   = NULL;
    double *d_mx_stdt = NULL;
    double *d_mx_van = NULL;
    int err_count = 0;
    int device_id = -1;

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, 0);
    if (err != hipSuccess) return -1;
    #if SHT_VERBOSE > 0
    printf("  cuda GPU \"%s\" found (warp size = %d, compute capabilities = %d.%d).\n", prop.name, prop.warpSize, prop.major, prop.minor);
    #endif
    if (prop.warpSize != WARPSZE) return -1;		// failure, SHTns requires a warpSize of 32.
    if (prop.major < 3) return -1;			// failure, SHTns requires compute cap. >= 3 (warp shuffle instructions)

    // Allocate the device input vector alm
    err = hipMalloc((void **)&d_alm, (2*nlm+THREADS_PER_BLOCK-1)*sizeof(double));	// allow some overflow.
    if (err != hipSuccess) err_count ++;
    if (shtns->mx_stdt) {
	// Allocate the device matrix for d(sin(t))/dt
	err = hipMalloc((void **)&d_mx_stdt, (2*nlm+THREADS_PER_BLOCK-1)*sizeof(double));
	if (err != hipSuccess) err_count ++;
	// Same thing for analysis
	err = hipMalloc((void **)&d_mx_van, (2*nlm+THREADS_PER_BLOCK-1)*sizeof(double));
	if (err != hipSuccess) err_count ++;
    }
    // Allocate the device input vector cos(theta) and gauss weights
    err = hipMalloc((void **)&d_ct, 2*nlat_2*sizeof(double));
    if (err != hipSuccess) err_count ++;
    // Allocate the device work vector qlm
    err = hipMalloc((void **)&d_qlm, 2*nlm*sizeof(double) * MAX_STRM);
    if (err != hipSuccess) err_count ++;
    // Allocate the device work vector q
    err = hipMalloc((void **)&d_q, shtns->nlat * shtns->nphi * sizeof(double) * MAX_STRM);
    if (err != hipSuccess) err_count ++;

    if (err_count == 0) {
	err = hipMemcpy(d_alm, shtns->alm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess)  err_count ++;
	if (shtns->mx_stdt) {
	    err = hipMemcpy(d_mx_stdt, shtns->mx_stdt, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	    if (err != hipSuccess)  err_count ++;
	    err = hipMemcpy(d_mx_van, shtns->mx_van, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	    if (err != hipSuccess)  err_count ++;
	}
	err = hipMemcpy(d_ct, shtns->ct, nlat_2*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess)  err_count ++;
	err = hipMemcpy(d_ct + nlat_2, shtns->wg, nlat_2*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess)  err_count ++;
    }
    
    for (int i=0; i<MAX_STRM; i++) {
	hipStreamCreate(&strm[i]);
    }

    /* cuFFT init */
    int nfft = shtns->nphi;
    if (nfft > 1) {
	// hipfftPlanMany(hipfftHandle *plan, int rank, int *n,   int *inembed, int istride, int idist,   int *onembed, int ostride, int odist,   hipfftType type, int batch);
	hipfftResult res;
	for (int i=0; i<MAX_STRM; i++) {
	    res = hipfftPlanMany((hipfftHandle*) &shtns->cufft_plan[i], 1, &nfft, &nfft, shtns->nlat_2, 1, &nfft, shtns->nlat_2, 1, HIPFFT_Z2Z, shtns->nlat_2);
	    if (res != HIPFFT_SUCCESS)  err_count ++;
	    hipfftSetStream(shtns->cufft_plan[i], strm[i]);
	}
    }

    if (err_count != 0) {
	hipFree(d_q);	hipFree(d_qlm);  hipFree(d_mx_van);	hipFree(d_mx_stdt);  hipFree(d_ct);  hipFree(d_alm);
	return -1;	// fail
    }

    shtns->d_alm = d_alm;
    shtns->d_ct  = d_ct;
    shtns->d_q   = d_q;
    shtns->d_qlm = d_qlm;
    shtns->d_mx_stdt = d_mx_stdt;
    shtns->d_mx_van = d_mx_van;
    hipGetDevice(&device_id);
    return device_id;		// success, return device_id
}

extern "C"
void cushtns_release_gpu(shtns_cfg shtns)
{
    hipfftDestroy(shtns->cufft_plan[0]);
    hipfftDestroy(shtns->cufft_plan[1]);
    hipStreamDestroy(strm[0]);
    hipStreamDestroy(strm[1]);
    if (shtns->d_q) hipFree(shtns->d_q);
    if (shtns->d_qlm) hipFree(shtns->d_qlm);
    if (shtns->d_ct) hipFree(shtns->d_ct);
    if (shtns->d_alm) hipFree(shtns->d_alm);
    if (shtns->d_mx_stdt) hipFree(shtns->d_mx_stdt);
    shtns->d_alm = 0;
}

/// \internal Enables parallel transforms on selected GPU device, if available. \see shtns_use_gpu 
extern "C"
int cushtns_use_gpu(int device_id)
{
    int count = 0;
    if (device_id >= 0) {
	hipGetDeviceCount(&count);
	if (count > 0) {
	    device_id = device_id % count;
	    hipSetDevice(device_id);
	    return device_id;
	}
    }
    return -1;		// disable gpu.
}


extern "C"
void SH_to_spat_gpu_hostfft(shtns_cfg shtns, cplx *Qlm, double *Vr, const long int llim)
{
    hipError_t err = hipSuccess;
    const int lmax = shtns->lmax;
    int mmax = shtns->mmax;
    const int mres = shtns->mres;
    const int nlm = shtns->nlm;
    const int nlat = shtns->nlat;
    const int nphi = shtns->nphi;
    double *d_alm = shtns->d_alm;
    double *d_ct = shtns->d_ct;

    // Launch the Legendre CUDA Kernel
    const int threadsPerBlock = THREADS_PER_BLOCK;	// can be from 32 to 1024, we should try to measure the fastest !
    const int blocksPerGrid =(nlat/2 + threadsPerBlock*NWAY - 1) / (threadsPerBlock*NWAY);
    double *d_qlm = shtns->d_qlm;
    double *d_q = shtns->d_q;
    if (llim < mmax*mres) mmax = llim / mres;	// truncate mmax too !
    if (mmax == 0) {
	double* Ql0;
	Ql0 = (double*) malloc((lmax+1)*sizeof(double));
	for (int l=0; l<=llim; l++) {
	    Ql0[l] = creal(Qlm[l]);
	}
	err = hipMemcpy(d_qlm, Ql0, (llim+1)*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess)  printf("failed copy qlm\n");

	leg_m0<0><<<blocksPerGrid, threadsPerBlock>>>(d_alm, d_ct, d_qlm, d_q, llim, nlat/2);
    } else {
	err = hipMemcpy(d_qlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess)  printf("failed copy qlm\n");

	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	if (llim <= SHT_L_RESCALE_FLY) {
	    leg_m_lowllim<0><<<blocks, threads, 2*threadsPerBlock*sizeof(double)>>>(d_alm, d_ct, d_qlm, d_q, llim, nlat/2, lmax,mres, nphi);
	} else {
	    leg_m_highllim<0><<<blocks, threads>>>(d_alm, d_ct, d_qlm, d_q, llim, nlat/2, lmax,mres, nphi);
	}
	// padd missing m's with 0 (m>mmax)
	if (2*(mmax+1) <= nphi)
	    hipMemset( d_q + (mmax+1)*nlat, 0, sizeof(double)*(nphi-2*mmax-1)*nlat );		// set to zero before fft
    }
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch cuda kernel (error code %s)!\n", hipGetErrorString(err));
    }

    err = hipMemcpy(Vr, d_q, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess)  printf("failed copy back : %s\n", hipGetErrorString(err));

    if (nphi > 1) {		// fft
  	if (shtns->fftc_mode >= 0) {
		if (shtns->fftc_mode == 0) {
		    fftw_execute_dft(shtns->ifftc, (cplx *) Vr, (cplx *) Vr);
		} else {		// split dft
		    printf("ERROR fft not supported\n");
		}
	}
    }
}

/// Perform SH transform on data that is already on the GPU. d_Qlm and d_Vr are pointers to GPU memory (obtained by hipMalloc() for instance)
template<int S>
void cuda_SH_to_spat(shtns_cfg shtns, cplx* d_Qlm, double *d_Vr, const long int llim, int strm_idx = 0)
{
    const int lmax = shtns->lmax;
    int mmax = shtns->mmax;
    const int mres = shtns->mres;
    const int nlat_2 = shtns->nlat_2;
    const int nphi = shtns->nphi;
    double *d_alm = shtns->d_alm;
    double *d_ct = shtns->d_ct;
    hipStream_t stream = strm[strm_idx];

    // Launch the Legendre CUDA Kernel
    const int threadsPerBlock = THREADS_PER_BLOCK;	// can be from 32 to 1024, we should try to measure the fastest !
    const int blocksPerGrid =(nlat_2 + threadsPerBlock - 1) / threadsPerBlock;
    if (nphi == 1) {
	leg_m0<S><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, (double*) d_Qlm, (double*) d_Vr, llim, nlat_2);
    } else {
	if (llim < mmax*mres) mmax = llim / mres;	// truncate mmax too !
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	if (mmax==0) {
	    leg_m0<S><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, (double*) d_Qlm, (double*) d_Vr, llim, nlat_2);
	} else
	if (llim <= SHT_L_RESCALE_FLY) {
	    leg_m_lowllim<S><<<blocks, threads, 2*threadsPerBlock*sizeof(double), stream>>>(d_alm, d_ct, (double*) d_Qlm, (double*) d_Vr, llim, nlat_2, lmax,mres, nphi);
	} else {
	    leg_m_highllim<S><<<blocks, threads, 0, stream>>>(d_alm, d_ct, (double*) d_Qlm, (double*) d_Vr, llim, nlat_2, lmax,mres, nphi);
	}
	// padd missing m's with 0 (m>mmax)
	if (2*(mmax+1) <= nphi)
	    hipMemsetAsync( d_Vr + (mmax+1)*2*nlat_2, 0, sizeof(double)*(nphi-2*mmax-1)*2*nlat_2, stream );		// set to zero before fft
	hipfftResult res;
	res = hipfftExecZ2Z((hipfftHandle) shtns->cufft_plan[strm_idx], (hipfftDoubleComplex*) d_Vr, (hipfftDoubleComplex*) d_Vr, HIPFFT_BACKWARD);
	if (res != HIPFFT_SUCCESS) printf("cufft error %d\n", res);
    }
}

extern "C"
void cu_SH_to_spat(shtns_cfg shtns, cplx* d_Qlm, double *d_Vr, const long int llim)
{
    cuda_SH_to_spat<0>(shtns, d_Qlm, d_Vr, llim);
}

extern "C"
void cu_SHsphtor_to_spat(shtns_cfg shtns, cplx* d_Slm, cplx* d_Tlm, double* d_Vt, double* d_Vp, const long llim)
{
    double* d_vwlm;
    const int nlm = shtns->nlm + (shtns->mmax+1);	// we need one more mode per m.
    const long nlm_stride = ((2*nlm+WARPSZE-1)/WARPSZE) * WARPSZE;
    hipError_t err = hipSuccess;

    // we need temporary storage here ...
    err = hipMalloc( (void **)&d_vwlm, (2*nlm_stride)*sizeof(double) );

    dim3 blocks((2*(shtns->lmax+2)+THREADS_PER_BLOCK-5)/(THREADS_PER_BLOCK-4), shtns->mmax+1);
    dim3 threads(THREADS_PER_BLOCK, 1);
    sphtor2scal_gpu <<<blocks, threads>>>
	(shtns->d_mx_stdt, (double*) d_Slm, (double*) d_Tlm, d_vwlm, d_vwlm+nlm_stride, llim, shtns->lmax, shtns->mres);
//    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) { printf("sphtor2scal_gpu error : %s!\n", hipGetErrorString(err));	return; }

    // SHT on the GPU
    cuda_SH_to_spat<1>(shtns, (cplx*) d_vwlm, d_Vt, llim+1, 0);
    cuda_SH_to_spat<1>(shtns, (cplx*) (d_vwlm + nlm_stride), d_Vp, llim+1, 1);

    hipFree(d_vwlm);
}

extern "C"
void cu_SHqst_to_spat(shtns_cfg shtns, cplx* d_Qlm, cplx* d_Slm, cplx* d_Tlm, double* d_Vr, double* d_Vt, double* d_Vp, const long llim)
{
    double* d_vwlm;
    const int nlm = shtns->nlm + (shtns->mmax+1);	// we need one more mode per m.
    const long nlm_stride = ((2*nlm+WARPSZE-1)/WARPSZE) * WARPSZE;
    hipError_t err = hipSuccess;

    cuda_SH_to_spat<0>(shtns, d_Qlm, d_Vr, llim, 0);	// scalar part on stream 0

    // we need temporary storage here ...
    err = hipMalloc( (void **)&d_vwlm, (2*nlm_stride)*sizeof(double) );

    dim3 blocks((2*(shtns->lmax+2)+THREADS_PER_BLOCK-5)/(THREADS_PER_BLOCK-4), shtns->mmax+1);
    dim3 threads(THREADS_PER_BLOCK, 1);
    sphtor2scal_gpu <<<blocks, threads, 0, strm[1]>>>
	(shtns->d_mx_stdt, (double*) d_Slm, (double*) d_Tlm, d_vwlm, d_vwlm+nlm_stride, llim, shtns->lmax, shtns->mres);
    hipDeviceSynchronize();		// make stream 2 wait for stream 1
    err = hipGetLastError();
    if (err != hipSuccess) { printf("sphtor2scal_gpu error : %s!\n", hipGetErrorString(err));	return; }

    // SHT on the GPU
    cuda_SH_to_spat<1>(shtns, (cplx*) d_vwlm, d_Vt, llim+1, 1);			// stream 1
    cuda_SH_to_spat<1>(shtns, (cplx*) (d_vwlm + nlm_stride), d_Vp, llim+1, 2);	// stream 2

    hipFree(d_vwlm);
}


/// Perform SH transform on data that is already on the GPU. d_Qlm and d_Vr are pointers to GPU memory (obtained by hipMalloc() for instance)
template<int S>
void cuda_spat_to_SH(shtns_cfg shtns, double *d_Vr, cplx* d_Qlm, const long int llim, int strm_idx = 0)
{
    const int lmax = shtns->lmax;
    int mmax = shtns->mmax;
    const int mres = shtns->mres;
    const int nlat_2 = shtns->nlat_2;
    const int nphi = shtns->nphi;
    const int nlm = shtns->nlm +S*(mmax+1);	// use more space for vector transform !!!
    double *d_alm = shtns->d_alm;
    double *d_ct = shtns->d_ct;
    hipStream_t stream = strm[strm_idx];

    // Launch the Legendre CUDA Kernel
    const int threadsPerBlock = THREADS_PER_BLOCK;	// can be from 32 to 1024, we should try to measure the fastest !
    const int blocksPerGrid =(nlat_2 + threadsPerBlock - 1) / (threadsPerBlock*NWAY);
    hipMemsetAsync(d_Qlm, 0, sizeof(double)*2*nlm, stream);		// set to zero before we start.
    if (nphi == 1) {
	ileg_m0<S><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, (double*) d_Vr, (double*) d_Qlm, llim, nlat_2);
    } else {
	hipfftResult res;
	res = hipfftExecZ2Z((hipfftHandle) shtns->cufft_plan[strm_idx], (hipfftDoubleComplex*) d_Vr, (hipfftDoubleComplex*) d_Vr, HIPFFT_BACKWARD);
	if (res != HIPFFT_SUCCESS) printf("cufft error %d\n", res);

	if (llim < mmax*mres) mmax = llim / mres;	// truncate mmax too !
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	if (mmax==0) {
	    ileg_m0<S><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, (double*) d_Vr, (double*) d_Qlm, llim, nlat_2);
	} else
	if (llim <= SHT_L_RESCALE_FLY) {
	    ileg_m_lowllim<S><<<blocks, threads, 0, stream>>>(d_alm, d_ct, (double*) d_Vr, (double*) d_Qlm, llim, nlat_2, lmax,mres,nphi);
	} else {
	    ileg_m_highllim<S><<<blocks, threads, 0, stream>>>(d_alm, d_ct, (double*) d_Vr, (double*) d_Qlm, llim, nlat_2, lmax,mres,nphi);
	}
    }
}

extern "C"
void cu_spat_to_SH(shtns_cfg shtns, double *d_Vr, cplx* d_Qlm, const long int llim)
{
    cuda_spat_to_SH<0>(shtns, d_Vr, d_Qlm, llim);
}


extern "C"
void SH_to_spat_gpu(shtns_cfg shtns, cplx *Qlm, double *Vr, const long int llim)
{
    hipError_t err = hipSuccess;
    const int nlm = shtns->nlm;
    const int nlat = shtns->nlat;
    const int nphi = shtns->nphi;

    double *d_qlm = shtns->d_qlm;
    double *d_q = shtns->d_q;

    // Allocate the device work vectors qlm and q
//    err = hipMalloc((void **)&d_qlm, ((2*nlm +31 + nlat*nphi+31)/32)*32*sizeof(double));
//    d_q = d_qlm + ((2*nlm+31)/32)*32;

    // copy spectral data to GPU
    err = hipMemcpy(d_qlm, Qlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("SH_to_spat_gpu failed copy qlm\n");	return; }

    // SHT on the GPU
    cuda_SH_to_spat<0>(shtns, (cplx*) d_qlm, d_q, llim);
    err = hipGetLastError();
    if (err != hipSuccess) { printf("SH_to_spat_gpu CUDA error : %s!\n", hipGetErrorString(err));	return; }

    // copy back spatial data
    err = hipMemcpy(Vr, d_q, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("SH_to_spat_gpu failed copy back: %s\n", hipGetErrorString(err));	return; }
    
//    hipFree(d_qlm);
}


/** \internal convert from vector SH to scalar SH
    Vlm =  st*d(Slm)/dtheta + I*m*Tlm
    Wlm = -st*d(Tlm)/dtheta + I*m*Slm
**/
void sphtor2scal(shtns_cfg shtns, cplx* Slm, cplx* Tlm, cplx* Vlm, cplx* Wlm, const int llim)
{
    const int mmax = shtns->mmax;
    const int lmax = shtns->lmax;
    const int mres = shtns->mres;
    for (int im=0; im<=mmax; im++) {
	const int m = im*mres;
	long l = (im*(2*(lmax+1)-(m+mres)))>>1;
	double* mx = shtns->mx_stdt + 2*l;
	cplx* Sl = (cplx*) &Slm[l];	// virtual pointer for l=0 and im
	cplx* Tl = (cplx*) &Tlm[l];
	cplx* Vl = (cplx*) &Vlm[l+im];
	cplx* Wl = (cplx*) &Wlm[l+im];
	const double em = m;
	
	cplx sl = Sl[m];
	cplx tl = Tl[m];
	cplx vs = 0.0;
	cplx wt = 0.0;
	for (int l=m; l<=llim; l++) {
	    double mxu = mx[2*l];
	    double mxl = mx[2*l+1];	// mxl for next l
	    vs += I*em*tl;
	    wt += I*em*sl;
	    cplx vs1 = mxl*sl;		// vs for next l
	    cplx wt1 = -mxl*tl;		// wt for next l
	    if (l<llim) {
		sl = Sl[l+1];		// kept for next l
		tl = Tl[l+1];
		vs += mxu*sl;
		wt -= mxu*tl;
	    }
	    Vl[l] = vs;
	    Wl[l] = wt;
	    vs = vs1;		wt = wt1;
	}
	Vl[llim+1] = vs;
	Wl[llim+1] = wt;
    }
}


extern "C"
void SHsphtor_to_spat_gpu(shtns_cfg shtns, cplx *Slm, cplx *Tlm, double *Vt, double *Vp, const long int llim)
{
    hipError_t err = hipSuccess;
    const int nlm = shtns->nlm;
    const int nlat = shtns->nlat;
    const int nphi = shtns->nphi;
    const int nlm2 = nlm + (shtns->mmax+1);	// one more data per m

    static double* d_vwlm = NULL;
    double* d_vtp;
//    static double* vw = NULL;

    const long nlm_stride = ((2*nlm2+WARPSZE-1)/WARPSZE) * WARPSZE;
    const long spat_stride = ((nlat*nphi+WARPSZE-1)/WARPSZE) * WARPSZE;

//    if (vw == NULL)
//	err = hipHostMalloc( (void**) &vw, ((nlm_stride > spat_stride) ? nlm_stride : spat_stride)*2*sizeof(double) );	// pinned buffer for transfer
    // Allocate the device work vectors
    if (d_vwlm == NULL)
    err = hipMalloc( (void **)&d_vwlm, (4*nlm_stride + 2*spat_stride)*sizeof(double) );
    d_vtp = d_vwlm + 4*nlm_stride;

/*   // convert on cpu & transfer (via pinned mem) 
    sphtor2scal(shtns, Slm, Tlm, (cplx*) vw, (cplx*) (vw + nlm_stride), llim);		// convert & copy to pinned mem
    err = hipMemcpy(d_vwlm, vw, 2*nlm_stride*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("failed copy vw\n");	return; }
*/
    // OR transfer and convert on gpu
    err = hipMemcpy(d_vwlm + 2*nlm_stride, Slm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("memcpy 1 error : %s!\n", hipGetErrorString(err));	return; }
    err = hipMemcpy(d_vwlm + 3*nlm_stride, Tlm, 2*nlm*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("memcpy 2 error : %s!\n", hipGetErrorString(err));	return; }
    dim3 blocks((2*(shtns->lmax+2)+THREADS_PER_BLOCK-5)/(THREADS_PER_BLOCK-4), shtns->mmax+1);
    dim3 threads(THREADS_PER_BLOCK, 1);
    sphtor2scal_gpu <<<blocks, threads>>>
	(shtns->d_mx_stdt, d_vwlm+2*nlm_stride, d_vwlm+3*nlm_stride, d_vwlm, d_vwlm+nlm_stride, llim, shtns->lmax, shtns->mres);
//    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) { printf("sphtor2scal_gpu error : %s!\n", hipGetErrorString(err));	return; }

    // SHT on the GPU
    cuda_SH_to_spat<1>(shtns, (cplx*) d_vwlm, d_vtp, llim+1, 0);
    cuda_SH_to_spat<1>(shtns, (cplx*) (d_vwlm + nlm_stride), d_vtp + spat_stride, llim+1, 1);
    err = hipGetLastError();
    if (err != hipSuccess) { printf("SH_to_spat CUDA error : %s!\n", hipGetErrorString(err));	return; }

    // copy back spatial data
    err = hipMemcpy(Vt, d_vtp, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(Vp, d_vtp + spat_stride, nlat*nphi*sizeof(double), hipMemcpyDeviceToHost);

/*	// OR copy to pinned memory first
    err = hipMemcpy(vw, d_vtp, 2*spat_stride*sizeof(double), hipMemcpyDeviceToHost);
    memcpy(Vt, vw, nlat*nphi*sizeof(double));
    memcpy(Vp, vw + spat_stride, nlat*nphi*sizeof(double));
*/

//    hipFree(d_vwlm);		d_vwlm = NULL;
//    hipHostFree(vw);
}



extern "C"
void spat_to_SH_gpu(shtns_cfg shtns, double *Vr, cplx *Qlm, const long int llim)
{
    hipError_t err = hipSuccess;
    const int nlm = shtns->nlm;
    const int nlat = shtns->nlat;
    const int nphi = shtns->nphi;

    double *d_qlm = shtns->d_qlm;
    double *d_q = shtns->d_q;

    // copy spatial data to GPU
    err = hipMemcpy(d_q, Vr, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("spat_to_SH_gpu failed copy q\n");	return; }

    // SHT on the GPU
    cu_spat_to_SH(shtns, d_q, (cplx*) d_qlm, llim);
    err = hipGetLastError();
    if (err != hipSuccess) { printf("spat_to_SH_gpu CUDA error : %s!\n", hipGetErrorString(err));	return; }

    // copy back spectral data
    err = hipMemcpy(Qlm, d_qlm, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("spat_to_SH_gpu failed copy back\n");	return; }
}


extern "C"
void spat_to_SHsphtor_gpu(shtns_cfg shtns, double *Vt, double *Vp, cplx *Slm, cplx *Tlm, const long int llim)
{
    hipError_t err = hipSuccess;
    const int nlm = shtns->nlm;
    const int nlat = shtns->nlat;
    const int nphi = shtns->nphi;
    const int nlm2 = nlm + (shtns->mmax+1);	// one more data per m

    static double* d_vwlm = NULL;
    double* d_vtp;
//    static double* vw = NULL;

    const long nlm_stride = ((2*nlm2+WARPSZE-1)/WARPSZE) * WARPSZE;
    const long spat_stride = ((nlat*nphi+WARPSZE-1)/WARPSZE) * WARPSZE;

//    if (vw == NULL)
//	err = hipHostMalloc( (void**) &vw, ((nlm_stride > spat_stride) ? nlm_stride : spat_stride)*2*sizeof(double) );	// pinned buffer for transfer
    // Allocate the device work vectors
    if (d_vwlm == NULL)
	 err = hipMalloc( (void **)&d_vwlm, (4*nlm_stride + 2*spat_stride)*sizeof(double) );
    d_vtp = d_vwlm + 4*nlm_stride;

    // copy spatial data to gpu
    err = hipMemcpy(d_vtp, Vt, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("memcpy 3 error : %s!\n", hipGetErrorString(err));	return; }
    err = hipMemcpy(d_vtp + spat_stride, Vp, nlat*nphi*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) { printf("memcpy 4 error : %s!\n", hipGetErrorString(err));	return; }

    // SHT on the GPU
    cuda_spat_to_SH<1>(shtns, d_vtp, (cplx*) d_vwlm, llim+1, 0);
    cuda_spat_to_SH<1>(shtns, d_vtp + spat_stride, (cplx*) (d_vwlm + nlm_stride), llim+1, 0);
    err = hipGetLastError();
    if (err != hipSuccess) { printf("spat_to_SHsphtor CUDA error : %s!\n", hipGetErrorString(err));	return; }

    hipDeviceSynchronize();

    dim3 blocks((2*(shtns->lmax+2)+THREADS_PER_BLOCK-5)/(THREADS_PER_BLOCK-4), shtns->mmax+1);
    dim3 threads(THREADS_PER_BLOCK, 1);
    scal2sphtor_gpu <<<blocks, threads>>>
	(shtns->d_mx_van, d_vwlm, d_vwlm+nlm_stride, d_vwlm+2*nlm_stride, d_vwlm+3*nlm_stride, llim, shtns->lmax, shtns->mres);
    err = hipGetLastError();
    if (err != hipSuccess) { printf("scal2sphtor_gpu error : %s!\n", hipGetErrorString(err));	return; }

    hipDeviceSynchronize();

    err = hipMemcpy(Slm, d_vwlm + 2*nlm_stride, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(Tlm, d_vwlm + 3*nlm_stride, 2*nlm*sizeof(double), hipMemcpyDeviceToHost);

//    hipFree(d_vwlm);		d_vwlm = NULL;
//    hipHostFree(vw);
}



extern "C"
void SH_to_spat_many_gpu(shtns_cfg shtns, int howmany, cplx *Qlm, double *Vr, const long int llim)
{
    hipError_t err = hipSuccess;
    hipEvent_t* event;
    const int nlm = shtns->nlm;
    const int lmax = shtns->lmax;
    int mmax = shtns->mmax;
    const int mres = shtns->mres;    
    const int nlat = shtns->nlat;
    const int nphi = shtns->nphi;
    const int nspat = shtns->nspat;
    double *d_alm = shtns->d_alm;
    double *d_ct = shtns->d_ct;

    double* pinned;

    double *d_qlm = shtns->d_qlm;
    double *d_q = shtns->d_q;

    const int nstreams = (howmany < MAX_STRM) ? howmany : MAX_STRM;

    size_t dist = (2*nlm > nspat) ? 2*nlm : nspat;		// largest size between input or output.
    dist = ((dist + 31) >> 5) << 5;	// round to 32*8 = 256 bytes.
    hipHostMalloc(&pinned, sizeof(double)*howmany*dist);	// alloc pinned memory for fast transfers.

    event = (hipEvent_t*) malloc(sizeof(hipEvent_t) * howmany);

    const int threadsPerBlock = THREADS_PER_BLOCK;	// can be from 32 to 1024, we should try to measure the fastest !
    const int blocksPerGrid =(nlat/2 + threadsPerBlock*NWAY - 1) / (threadsPerBlock*NWAY);

    // copy data and launch kernels in multiple concurrent streams.
    for (int k = 0; k < howmany; k++) {
	const hipStream_t stream = strm[k % MAX_STRM];
	memcpy(pinned + k*dist,  Qlm + (k%MAX_STRM)*nlm, 2*nlm*sizeof(double));	// copy to pinned mem
	hipMemcpyAsync(d_qlm + (k%MAX_STRM)*2*nlm, pinned + k*dist, 2*nlm*sizeof(double), hipMemcpyHostToDevice, stream);		// data transfer to gpu
	dim3 blocks(blocksPerGrid, mmax+1);
	dim3 threads(threadsPerBlock, 1);
	if (mmax==0) {
	    leg_m0<0><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_alm, d_ct, d_qlm + (k%MAX_STRM)*2*nlm, d_q + (k%MAX_STRM)*nspat, llim, nlat/2);
	} else if (llim <= SHT_L_RESCALE_FLY) {
	    leg_m_lowllim<0><<<blocks, threads, 2*threadsPerBlock*sizeof(double), stream>>>(d_alm, d_ct, d_qlm + (k%MAX_STRM)*2*nlm, d_q + (k%MAX_STRM)*nspat, llim, nlat/2, lmax,mres, nphi);
	} else {
	    leg_m_highllim<0><<<blocks, threads, 0, stream>>>(d_alm, d_ct, d_qlm + (k%MAX_STRM)*2*nlm, d_q + (k%MAX_STRM)*nspat, llim, nlat/2, lmax,mres, nphi);
	}
	// padd missing m's with 0 (m>mmax)
	if (2*(mmax+1) <= nphi)
	    hipMemsetAsync( d_q + (k%MAX_STRM)*nspat + (mmax+1)*nlat, 0, sizeof(double)*(nphi-2*mmax-1)*nlat, stream );		// set to zero before fft

	hipMemcpyAsync(pinned + k*dist, d_q + (k%MAX_STRM)*nspat, nspat*sizeof(double), hipMemcpyDeviceToHost, stream);		// transfer back
	hipEventCreateWithFlags(event + k,  hipEventBlockingSync | hipEventDisableTiming);
	hipEventRecord(event[k], stream);
    }

    // wait for event completion, and perform fft (out-of-place) by the CPU (possibly multi-threaded).
    for (int k = 0; k < howmany; k++) {
	hipEventSynchronize( event[k] );		// wait for event
	hipEventDestroy(event[k]);			// get rid of event
	if (nphi > 1) {		// fft
	    if (shtns->fftc_mode == 0) {
		memcpy(Vr + (k%MAX_STRM)*nspat, pinned + k*dist, nspat*sizeof(double));	// copy from pinned mem (should be replaced by oop fft)
		fftw_execute_dft(shtns->ifftc, (cplx *) (Vr + (k%MAX_STRM)*nspat), (cplx *) (Vr + (k%MAX_STRM)*nspat));
		//fftw_execute_dft(shtns->ifftc, (cplx *) (pinned + k*dist), (cplx *) (Vr + k*nspat));	// oop
	    } else {		// split dft
		printf("ERROR fft not supported\n");
	    }
	}
    }

    free(event);
    hipHostFree(pinned);
}

